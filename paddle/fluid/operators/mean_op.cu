#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/operators/mean_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T>
struct DivideFunctor {
  HOSTDEVICE explicit inline DivideFunctor(int n)
      : n_inv(static_cast<T>(1.0 / n)) {}

  HOSTDEVICE inline T operator()(const T& x) const { return x * n_inv; }

 private:
  T n_inv;
};
template <typename T>
__global__ void MeanAvgKernel(const T* in_data, const size_t num, T* out_data) {
  double val = 0.0;
  for (size_t i = 0; i < num; ++i) {
    val += static_cast<double>(in_data[i]);
  }
  out_data[0] = val / static_cast<double>(num);
}
template <typename T>
__global__ void MeanRunKernel(const T* in_data, T* out_data, int N) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  T data = in_data[0];
  for (; idx < N; idx += blockDim.x * gridDim.x) {
    out_data[idx] = data / (static_cast<T>(N));
  }
}

template <typename DeviceContext, typename T>
class MeanCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* input = context.Input<Tensor>("X");
    auto* output = context.Output<Tensor>("Out");

    output->mutable_data<T>(context.GetPlace());
    auto size_prob = input->numel();
    const T* in_data = input->data<T>();
    T* out_data = output->mutable_data<T>(context.GetPlace());
    auto stream = context.cuda_device_context().stream();

    MeanAvgKernel<T><<<1, 1, 0, stream>>>(in_data, size_prob, out_data);

    //    DivideFunctor<T> transformer(size_prob);
    //    hipcub::TransformInputIterator<T, DivideFunctor<T>, const T*> trans_x(
    //        in_data, transformer);
    //    size_t temp_storage_bytes = 0;
    //
    //    auto err = hipcub::DeviceReduce::Sum(nullptr, temp_storage_bytes,
    //    trans_x,
    //                                      out_data, size_prob, stream);
    //    PADDLE_ENFORCE_CUDA_SUCCESS(err);
    //    framework::Tensor tmp;
    //    auto* temp_storage = tmp.mutable_data<uint8_t>(
    //        framework::make_ddim({static_cast<int64_t>(temp_storage_bytes)}),
    //        context.GetPlace());
    //    err = hipcub::DeviceReduce::Sum(temp_storage, temp_storage_bytes,
    //    trans_x,
    //                                 out_data, size_prob, stream);
    //    PADDLE_ENFORCE_CUDA_SUCCESS(err);
  }
};

template <typename DeviceContext, typename T>
class MeanCUDAGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto OG = context.Input<Tensor>(framework::GradVarName("Out"));
    PADDLE_ENFORCE_EQ(OG->numel(), 1,
                      platform::errors::InvalidArgument(
                          "Mean Gradient Input Tensor len should be 1. But "
                          "received Out@Grad's elements num is %d.",
                          OG->numel()));
    auto IG = context.Output<Tensor>(framework::GradVarName("X"));
    IG->mutable_data<T>(context.GetPlace());

    auto in_data = OG->data<T>();
    auto size_prob = IG->numel();
    auto out_data = IG->data<T>();
    int threads = 512;
    int grid = (size_prob + threads - 1) / threads;
    auto stream = context.cuda_device_context().stream();
    MeanRunKernel<T><<<grid, threads, 0, stream>>>(in_data, out_data,
                                                   size_prob);
  }
};

template <typename T>
__global__ void MaskMeanAvgKernel(const size_t N, const T* in_data,
                                  const T* mask_data, T* out_num, T* out_data) {
  double val = 0.0;
  double num = 0.0;
  for (size_t i = 0; i < N; ++i) {
    val += static_cast<double>(in_data[i] * mask_data[i]);
    num += static_cast<double>(mask_data[i]);
  }
  if (num > 0.0) {
    out_data[0] = static_cast<T>(val / num);
    out_num[0] = static_cast<T>(num);
  } else {
    out_data[0] = 0.0;
    out_num[0] = 0.0;
  }
}

template <typename DeviceContext, typename T>
class MaskMeanCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* input = context.Input<Tensor>("X");
    auto* mask = context.Input<Tensor>("Mask");
    auto* output = context.Output<Tensor>("Out");
    auto* num = context.Output<Tensor>("Num");

    T* out_data = output->mutable_data<T>(context.GetPlace());
    T* out_num = num->mutable_data<T>(context.GetPlace());

    auto size_prob = input->numel();
    const T* in_data = input->data<T>();
    const T* mask_data = mask->data<T>();
    auto stream = context.cuda_device_context().stream();

    MaskMeanAvgKernel<T><<<1, 1, 0, stream>>>(size_prob, in_data, mask_data,
                                              out_num, out_data);
  }
};

template <typename T>
__global__ void MaskRunKernel(const size_t N, const T* in_data,
                              const T* mask_data, const T* mask_num,
                              T* out_data) {
  size_t idx = blockDim.x * blockIdx.x + threadIdx.x;

  const T& num = mask_num[0];
  if (num > static_cast<T>(0.0)) {
    const T& val = in_data[0] / num;
    for (; idx < N; idx += blockDim.x * gridDim.x) {
      out_data[idx] = val * mask_data[idx];
    }
  } else {
    for (; idx < N; idx += blockDim.x * gridDim.x) {
      out_data[idx] = static_cast<T>(0.0);
    }
  }
}

template <typename DeviceContext, typename T>
class MaskMeanCUDAGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto mask = context.Input<Tensor>("Mask");
    auto num = context.Input<Tensor>("Num");
    auto OG = context.Input<Tensor>(framework::GradVarName("Out"));
    PADDLE_ENFORCE_EQ(OG->numel(), 1,
                      platform::errors::InvalidArgument(
                          "Mean Gradient Input Tensor len should be 1. But "
                          "received Out@Grad's elements num is %d.",
                          OG->numel()));
    auto IG = context.Output<Tensor>(framework::GradVarName("X"));

    auto in_data = OG->data<T>();
    auto size_prob = IG->numel();
    auto out_data = IG->mutable_data<T>(context.GetPlace());
    auto num_data = num->data<T>();
    int threads = 512;
    int grid = (size_prob + threads - 1) / threads;
    auto stream = context.cuda_device_context().stream();
    MaskRunKernel<T><<<grid, threads, 0, stream>>>(
        size_prob, in_data, mask->data<T>(), num_data, out_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    mean, ops::MeanCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::MeanCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::MeanCUDAKernel<paddle::platform::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    mean_grad,
    ops::MeanCUDAGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::MeanCUDAGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::MeanCUDAGradKernel<paddle::platform::CUDADeviceContext,
                            plat::float16>);
// mask mean
REGISTER_OP_CUDA_KERNEL(
    mask_mean,
    ops::MaskMeanCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::MaskMeanCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::MaskMeanCUDAKernel<paddle::platform::CUDADeviceContext,
                            plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    mask_mean_grad,
    ops::MaskMeanCUDAGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::MaskMeanCUDAGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::MaskMeanCUDAGradKernel<paddle::platform::CUDADeviceContext,
                                plat::float16>);
